#include <hip/hip_runtime.h>

#include <iostream>

template <typename C, typename T>
__global__
void awkward_listarray_compact_offsets(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
	tooffsets[0] = 0;
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i == 0) {
		for (int idx = 0; idx < length; idx++) {
			C start = fromstarts[startsoffset + idx];
			C stop = fromstops[stopsoffset + idx];
			assert(start < stop);
			tooffsets[idx + 1] = tooffsets[idx] + (stop - start);
		}
	}
}

int main() {
	int tooffsets[6];
	int fromstarts[] = { 11, 12, 13, 14, 15 };
	int fromstops[] = { 21, 22, 23, 24, 25 };
	int* d_tooffsets, * d_fromstarts, * d_fromstops;
	hipMalloc((void**)&d_tooffsets, 6*sizeof(int));
	//cudaMemcpy(d_tooffsets, tooffsets, 6 * sizeof(int), cudaMemcpyHostToDevice);
	hipMalloc((void**)&d_fromstarts, 5 * sizeof(int));
	hipMemcpy(d_fromstarts, fromstarts, 5 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_fromstops, 5 * sizeof(int));
	hipMemcpy(d_fromstops, fromstops, 5 * sizeof(int), hipMemcpyHostToDevice);
	awkward_listarray_compact_offsets <int, int><<<1, 5>>> (d_tooffsets, d_fromstarts, d_fromstops, 0, 0, 5);
	//cudaDeviceSynchronize();
	hipMemcpy(tooffsets, d_tooffsets, 6 * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_tooffsets);
	hipFree(d_fromstarts);
	hipFree(d_fromstops);
	for (int i = 0; i < 6; i++) {
		std::cout << tooffsets[i] << "\n";
	}
	return 0;
}