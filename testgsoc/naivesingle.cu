#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>

template <typename C, typename T>
__global__
void awkward_listarray_compact_offsets(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx == 0) tooffsets[0] = 0;
  if (idx < length) {
    if (idx == 0) {
      for (int i = 0; i < length; i++) {
        C start = fromstarts[startsoffset + i];
        C stop = fromstops[stopsoffset + i];
        assert(start < stop);
        tooffsets[i + 1] = tooffsets[i] + (stop - start);
      }
    }
  }
}

template <typename T, typename C>
void offload(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
  int* d_tooffsets, * d_fromstarts, * d_fromstops;
  hipMalloc((void**)&d_tooffsets, (length + 1) * sizeof(int));
  hipMalloc((void**)&d_fromstarts, length * sizeof(int));
  hipMemcpy(d_fromstarts, fromstarts, length * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_fromstops, length * sizeof(int));
  hipMemcpy(d_fromstops, fromstops, length * sizeof(int), hipMemcpyHostToDevice);
  int block, thread;
  if (length > 1024) {
    block = (length / 1024) + 1;
    thread = 1024;
  }
  else {
    thread = length;
    block = 1;
  }
  awkward_listarray_compact_offsets <int, int> << <block, thread >> > (d_tooffsets, d_fromstarts, d_fromstops, startsoffset, stopsoffset, length);
  hipDeviceSynchronize();
  hipMemcpy(tooffsets, d_tooffsets, (length + 1) * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_tooffsets);
  hipFree(d_fromstarts);
  hipFree(d_fromstops);
}

int main() {
  const int size = 10;
  int tooffsets[size + 1], fromstarts[size], fromstops[size];
  for (int i = 0; i < size; i++) {
    fromstarts[i] = i;
    fromstops[i] = i + 10;
  }
  offload<int, int>(tooffsets, fromstarts, fromstops, 0, 0, size);
  for (int i = 0; i < size + 1; i++) {
    std::cout << tooffsets[i] << "\n";
  }
  return 0;
}