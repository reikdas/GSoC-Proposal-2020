#include <hip/hip_runtime.h>

#include <iostream>

template <typename T, typename C>
__global__
void awkward_listarray_compact_offsets(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length, T* sums) {
  int thid = threadIdx.x + (blockIdx.x * blockDim.x);
  extern __shared__ T temp[];
  int pout = 0, pin = 1;
  if (thid < length) {
    temp[thid] = fromstops[stopsoffset + thid] - fromstarts[startsoffset + thid];
    __syncthreads();
    for (int offset = 1; offset < length; offset *=2) {
      pout = 1 - pout;
      pin = 1 - pout;
      if (thid >= offset)
        temp[pout*length + thid] = temp[pin*length + thid - offset] + temp[pin*length + thid];
      else
        temp[pout*length + thid] = temp[pin*length + thid];
      __syncthreads();
    }
    tooffsets[thid] = temp[pout*length + thid];
    __syncthreads();
    if ((thid == 1023) || ((blockIdx.x != 0) && (thid == ((1024 * (blockIdx.x + 1))-1))) || (thid == length-1)) {
        sums[blockIdx.x] = tooffsets[thid];
    }
  }
}

template <typename T>
__global__
void prefix_sum(T* tooffsets, int length) {
  int thid = threadIdx.x + (blockIdx.x * blockDim.x);
  extern __shared__ T temp[];
  int pout = 0, pin = 1;
  if (thid < length) {
    temp[thid] = tooffsets[thid];
    __syncthreads();
    for (int offset = 1; offset < length; offset *=2) {
      pout = 1 - pout;
      pin = 1 - pout;
      if (thid >= offset)
        temp[pout*length + thid] = temp[pin*length + thid - offset] + temp[pin*length + thid];
      else
        temp[pout*length + thid] = temp[pin*length + thid];
      __syncthreads();
    }
    tooffsets[thid] = temp[pout*length + thid];
  }
}

template<typename T>
__global__
void adder(T* tooffsets, T* sums, int64_t length) {
  int thid = threadIdx.x + (blockIdx.x * blockDim.x);
  if (blockIdx.x != 0 && thid < length)
    tooffsets[thid] += sums[blockIdx.x - 1];
}

template <typename T, typename C>
void offload(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
  T* d_tooffsets, * d_sums;
  C* d_fromstarts, * d_fromstops;
  hipMalloc((void**)&d_tooffsets, (length+1) * sizeof(T));
  hipMalloc((void**)&d_fromstarts, length * sizeof(C));
  hipMemcpy(d_fromstarts, fromstarts, length * sizeof(C), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_fromstops, length * sizeof(C));
  hipMemcpy(d_fromstops, fromstops, length * sizeof(C), hipMemcpyHostToDevice);
  int block, thread;
  if (length > 1024) {
    block = (length / 1024) + 1;
    thread = 1024;
  }
  else {
    thread = length;
    block = 1;
  }
  hipMalloc((void**)&d_sums, block*sizeof(T));
  awkward_listarray_compact_offsets<T, C><<<block, thread, length*2*sizeof(T)>>>(d_tooffsets, d_fromstarts, d_fromstops, startsoffset, stopsoffset, length, d_sums);
  hipDeviceSynchronize();
  prefix_sum<T><<<1, block, block*2*sizeof(T)>>>(d_sums, block);
  hipDeviceSynchronize();
  adder<T><<<block, thread>>>(d_tooffsets, d_sums, length);
  hipDeviceSynchronize();
  hipMemcpy(tooffsets, d_tooffsets, (length + 1) * sizeof(T), hipMemcpyDeviceToHost);
  tooffsets[length] = tooffsets[length - 1] + fromstops[length - 1 + stopsoffset] - fromstarts[length - 1 + startsoffset];
  hipFree(d_tooffsets);
  hipFree(d_fromstarts);
  hipFree(d_fromstops);
  hipFree(d_sums);
}

int main() {
  const int size = 6000;
  int tooffsets[size + 1], fromstarts[size], fromstops[size];
  for (int i = 0; i < size; i++) {
    fromstarts[i] = i;
    fromstops[i] = i + 10;
  }
  offload<int, int>(tooffsets, fromstarts, fromstops, 0, 0, size);
  for (int i = 0; i < size + 1; i++) {
	  std::cout << tooffsets[i] << "\n";
  }
  return 0;
}