#include <hip/hip_runtime.h>

#include <iostream>

__global__
void awkward_listarray_compact_offsets(int* tooffsets, const int* fromstarts, const int* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
	tooffsets[0] = 0;
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
	if (i == 0) {
		for (int idx = 0; idx < length; idx++) {
			int start = fromstarts[startsoffset + idx];
			int stop = fromstops[stopsoffset + idx];
			if (stop < start) {
			}
			else {
				tooffsets[idx + 1] = tooffsets[idx] + (stop - start);
			}
		}
	}
	__syncthreads();
}

int main() {
	int tooffsets[6];
	int fromstarts[] = { 11, 12, 13, 14, 15 };
	int fromstops[] = { 21, 22, 23, 24, 25 };
	int* d_tooffsets, * d_fromstarts, * d_fromstops;
	hipMalloc((void**)&d_tooffsets, 6*sizeof(int));
	//cudaMemcpy(d_tooffsets, tooffsets, 6 * sizeof(int), cudaMemcpyHostToDevice);
	hipMalloc((void**)&d_fromstarts, 5 * sizeof(int));
	hipMemcpy(d_fromstarts, fromstarts, 5 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_fromstops, 5 * sizeof(int));
	hipMemcpy(d_fromstops, fromstops, 5 * sizeof(int), hipMemcpyHostToDevice);
	awkward_listarray_compact_offsets <<<1, 5>>> (d_tooffsets, d_fromstarts, d_fromstops, 0, 0, 5);
	hipDeviceSynchronize();
	hipMemcpy(tooffsets, d_tooffsets, 6 * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_tooffsets);
	hipFree(d_fromstarts);
	hipFree(d_fromstops);
	for (int i = 0; i < 6; i++) {
		std::cout << tooffsets[i] << "\n";
	}
	return 0;
}