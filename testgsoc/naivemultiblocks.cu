#include <hip/hip_runtime.h>

#include <iostream>
#include <algorithm>

// Only single thread block
template <typename C, typename T>
__global__
void awkward_listarray_compact_offsets(int block, T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
	int idx = threadIdx.x + (block*1024);
  if (idx < length) {
    if (idx == 0) tooffsets[0] = 0;
    for (int i = block*1024; i < std::min((int)length, (block+1)*1024); i++) {
      __syncthreads();
      if (i == idx) {
        C start = fromstarts[startsoffset + i];
        C stop = fromstops[stopsoffset + i];
        assert(start < stop);
        tooffsets[i + 1] = tooffsets[i] + (stop - start);
      }
    }
  }
}

template <typename T, typename C>
void offload(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
  int* d_tooffsets, * d_fromstarts, * d_fromstops;
  hipMalloc((void**)&d_tooffsets, (length+1) * sizeof(int));
  hipMalloc((void**)&d_fromstarts, length * sizeof(int));
  hipMemcpy(d_fromstarts, fromstarts, length * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_fromstops, length * sizeof(int));
  hipMemcpy(d_fromstops, fromstops, length * sizeof(int), hipMemcpyHostToDevice);
  int block, thread;
  if (length > 1024) {
    block = (length / 1024) + 1;
    thread = 1024;
  }
  else {
    thread = length;
    block = 1;
  }
  for (int i=0; i<block; i++) {
    awkward_listarray_compact_offsets <int, int><<<1, thread>>>(i, d_tooffsets, d_fromstarts, d_fromstops, startsoffset, stopsoffset, length);
  }
  hipDeviceSynchronize();
  hipMemcpy(tooffsets, d_tooffsets, (length + 1) * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_tooffsets);
  hipFree(d_fromstarts);
  hipFree(d_fromstops);
}

int main() {
  const int size = 10000;
  int tooffsets[size + 1], fromstarts[size], fromstops[size];
  for (int i = 0; i < size; i++) {
    fromstarts[i] = i;
    fromstops[i] = i + 10;
  }
  offload<int, int>(tooffsets, fromstarts, fromstops, 0, 0, size);
  for (int i = 0; i < size + 1; i++) {
	  std::cout << tooffsets[i] << "\n";
  }
  return 0;
}
