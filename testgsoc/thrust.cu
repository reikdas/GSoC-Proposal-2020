#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <hip/hip_runtime.h>
#include ""
#include <assert.h>

template <typename T, typename C>
__global__
void sub(T* output, const C* starter, const C* stopper, int64_t startsoffset, int64_t stopsoffset, int64_t n) {
  int thid = threadIdx.x + blockIdx.x * blockDim.x;
  if (thid < n) {
    C start = starter[thid + startsoffset];
    C stop = stopper[thid + stopsoffset];
    assert(start <= stop);
    output[thid] = stop - start;
  }
}

template <typename T, typename C>
void prefix_sum(T* output, const C* arr, const C* arr2, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
  int block, thread;
  if (length > 1024) {
    block = (length / 1024) + 1;
    thread = 1024;
  }
  else {
    thread = length;
    block = 1;
  }
  T* d_output;
  C* d_arr, * d_arr2;
  hipMalloc((void**)&d_output, length * sizeof(T));
  hipMalloc((void**)&d_arr, length * sizeof(C));
  hipMemcpy(d_arr, arr, length * sizeof(C), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_arr2, length * sizeof(C));
  hipMemcpy(d_arr2, arr2, length * sizeof(C), hipMemcpyHostToDevice);
  sub<T, C><<<block, thread>>>(d_output, d_arr, d_arr2, startsoffset, stopsoffset, length);
  hipDeviceSynchronize();
  thrust::device_vector<T> data(d_output, d_output+length);
  thrust::device_vector<T> temp(data.size() + 1);
  thrust::exclusive_scan(data.begin(), data.end(), temp.begin());
  temp[data.size()] = data.back() + temp[data.size() - 1];
  thrust::copy(temp.begin(), temp.end(), output);
  hipFree(d_output);
  hipFree(d_arr);
  hipFree(d_arr2);
}

int main() {
  int const size = 100000;
  int starter[size], stopper[size], output[size + 1];
  for (int i = 0; i < size; i++) {
    starter[i] = i;
    stopper[i] = i + 1;
  }
  prefix_sum<int, int>(output, starter, stopper, 0, 0, size);
  hipDeviceSynchronize();
  for (int i = 0; i < size + 1; i++) {
    std::cout << output[i] << "\n";
  }
}
