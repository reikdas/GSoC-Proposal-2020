#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <hip/hip_runtime.h>
#include ""

template <typename T>
void prefix_sum(T* output, const T* arr, const T* arr2, const int size) {
  thrust::device_vector<T> d_arr(arr, arr + size);
  thrust::device_vector<T> d_arr2(arr2, arr2 + size);
  thrust::device_vector<T> data(size);
  thrust::transform(d_arr2.begin(), d_arr2.end(), d_arr.begin(), data.begin(), thrust::minus<T>());
  thrust::device_vector<T> temp(data.size() + 1);
  thrust::exclusive_scan(data.begin(), data.end(), temp.begin());
  temp[data.size()] = data.back() + temp[data.size() - 1];
  thrust::copy(temp.begin(), temp.end(), output);
}

int main() {
  int const size = 70000;
  int starter[size], stopper[size], output[size + 1];
  for (int i = 0; i < size; i++) {
    starter[i] = i;
    stopper[i] = i + 1;
  }
  prefix_sum<int>(output, starter, stopper, size);
  for (int i = 0; i < size + 1; i++) {
    std::cout << output[i] << "\n";
  }
}