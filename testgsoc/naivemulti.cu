#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>

template <typename C, typename T>
__global__
void awkward_listarray_compact_offsets(T* tooffsets, const C* fromstarts, const C* fromstops, int64_t startsoffset, int64_t stopsoffset, int64_t length) {
	tooffsets[0] = 0;
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	for (int i = 0; i < length; i++) {
		if (i == idx) {
			C start = fromstarts[startsoffset + i];
			C stop = fromstops[stopsoffset + i];
			assert(start < stop);
			tooffsets[i + 1] = tooffsets[i] + (stop - start);
		}
		__syncthreads();
	}
}

int main() {
	int tooffsets[6];
	int fromstarts[] = { 11, 12, 13, 14, 15 };
	int fromstops[] = { 21, 22, 23, 24, 25 };
	int* d_tooffsets, * d_fromstarts, * d_fromstops;
	hipMalloc((void**)&d_tooffsets, 6 * sizeof(int));
	//hipMemcpy(d_tooffsets, tooffsets, 6 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_fromstarts, 5 * sizeof(int));
	hipMemcpy(d_fromstarts, fromstarts, 5 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_fromstops, 5 * sizeof(int));
	hipMemcpy(d_fromstops, fromstops, 5 * sizeof(int), hipMemcpyHostToDevice);
	awkward_listarray_compact_offsets <int, int> << <1, 5 >> > (d_tooffsets, d_fromstarts, d_fromstops, 0, 0, 5);
	//hipDeviceSynchronize();
	hipMemcpy(tooffsets, d_tooffsets, 6 * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_tooffsets);
	hipFree(d_fromstarts);
	hipFree(d_fromstops);
	for (int i = 0; i < 6; i++) {
		std::cout << tooffsets[i] << "\n";
	}
	return 0;
}